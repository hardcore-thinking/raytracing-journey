#include <iostream>
#include <cstdlib>

#include "hip/hip_runtime.h"


__global__ void HelloGPU() {
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int tz = threadIdx.z;

	unsigned int bix = blockIdx.x;
	unsigned int biy = blockIdx.y;
	unsigned int biz = blockIdx.z;

	unsigned int bdx = blockDim.x;
	unsigned int bdy = blockDim.y;
	unsigned int bdz = blockDim.z;

	printf("Currently running:\n"
		   " > threadIdx  = (x: %4u, y: %4u, z: %4u)\n"
		   " > blockIdx   = (x: %4u, y: %4u, z: %4u)\n"
		   " > blockDim   = (x: %4u, y: %4u, z: %4u)\n", tx, ty, tz, bix, biy, biz, bdx, bdy, bdz);
}

int main() {
	HelloGPU<<<1, 16>>>();

	hipDeviceSynchronize();

	return EXIT_SUCCESS;
}