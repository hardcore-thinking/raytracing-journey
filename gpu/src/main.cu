#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""

// Single-precision a * x + y
__global__ void SAXPY(int n, float a, float* x, float* y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < n) {
		y[i] = a * x[i] + y[i];
	}
}

int main() {
	int N = 1 << 20;

	float* x = (float*) malloc(N * sizeof(float));
	if (x == nullptr) {
		std::cerr << "[ERROR] Unable to allocate memory for the x vector on the host memory." << std::endl;
		goto exit;
	}

	float* y = (float*) malloc(N * sizeof(float));
	if (y == nullptr) {
		std::cerr << "[ERROR] Unable to allocate memory for the y vector on the host memory." << std::endl;
		goto exit;
	}
	
	float* d_x = nullptr;
	float* d_y = nullptr;

	if (hipMalloc(&d_x, N * sizeof(float)) != hipSuccess) {
		std::cerr << "[ERROR] Unable to allocate memory for the d_y vector on the device memory." << std::endl;
		goto exit;
	}

	if (hipMalloc(&d_y, N * sizeof(float)) != hipSuccess) {
		std::cerr << "[ERROR] Unable to allocate memory for the d_y vector on the device memory." << std::endl;
		goto exit;
	}

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	if (hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "[ERROR] Unable to allocate memory for the y vector." << std::endl;
		goto exit;
	}

	if (hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		std::cerr << "[ERROR] Unable to allocate memory for the y vector." << std::endl;
		goto exit;
	}

	SAXPY <<<(N + 255) / 256, 256>>> (N, 2.0f, d_x, d_y);

	if (hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cerr << "[ERROR] Unable to allocate memory for the y vector." << std::endl;
		goto exit;
	}

	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = std::max(maxError, std::abs(y[i] - 4.0f));
	}

	std::cout << "Max error: " << maxError << std::endl;

	exit:
		if (x != nullptr) {
			free(x);
		}

		if (y != nullptr) {
			free(y);
		}

		if (d_x != nullptr) {
			hipFree(d_x);
		}

		if (d_y != nullptr) {
			hipFree(d_y);
		}

	return EXIT_SUCCESS;
}